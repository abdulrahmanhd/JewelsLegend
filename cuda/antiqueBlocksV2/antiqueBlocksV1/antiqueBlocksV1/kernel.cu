#include "hip/hip_runtime.h"
#include <windows.h>
#include <iostream>
#include <string>
#include "block.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include "hip/device_functions.h"
using namespace std;


const int nColores = 9;


////Metodos GPU
__device__ Block obtenerBloqueSiguiente(Block bloque[], Block bloqueActual, int tx, int ty, int filas, int columnas, int colorInit, bool bloqueElegido);
__device__ int bloquesIgualesAdyacentes(Block bloque[], int filas, int columnas, int x, int y);



int colocarArriba(Block *bloques, int i, int j, int filas, int columnas);
int colocarDerecha(Block *bloques, int i, int j, int filas, int columnas);
int moveBlocks(Block *bloques, int filas, int columnas);
Block *explotarBomba(Block *bloques, int posX, int posY, int filas, int columnas);

//Metodo Kernel. Llamado por la CPU y ejecutado por el Dispositivo
__global__ void bloquesKernel(Block *dev_bloques, Block *dev_bloquesSiguientes, int filas, int columnas, int miX, int miY) {
	
	//Hilos en 2D

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	bool bomba = false;
	
	if(tx == miX && ty == miY){
		printf("Se explotar� el bloque: %d, %d\n", miX, miY);

	
	}


	
	
	if((tx == miX && ty == miY) || (tx == miX -1 && ty == miY -1) || (tx == miX  && ty == miY -1) || (tx == miX -1 && ty == miY ) || (tx == miX +1 && ty == miY +1) || (tx == miX && ty == miY +1) || (tx == miX +1 && ty == miY ))
	{
		Block bloqueActual = dev_bloques[tx*filas + ty];

		Block bloqueSiguiente = obtenerBloqueSiguiente(dev_bloques, bloqueActual, tx, ty, filas, columnas, dev_bloques[miX*filas + miY].color, bomba);

		dev_bloquesSiguientes[tx*filas + ty] = bloqueSiguiente;
	}
	else
	{

		Block bloqueActual = dev_bloques[tx*filas + ty];

		dev_bloquesSiguientes[tx*filas + ty] = bloqueActual;

	}

}


__device__ int bloquesIgualesAdyacentes(Block* bloques, int filas, int columnas, int x, int y) {
	int bloquesAdyacentes = 0;
	int posX = x, posY = y;  
	 

	//ARRIBA-IZQU
	x = (posX - 1) % filas;
	y = (posY - 1) % columnas;
	if (posX == 0)
		x = filas - 1;
	if (posY == 0)
		y = columnas - 1;
	if (bloques[x*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;
		
	}

	//ARRIBA 
	x = (posX - 1);
	if (posX == 0)
		x = filas - 1;
	if (bloques[x*filas + posY].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

		
	}

	//ARRIBA-DERECHA
	x = (posX - 1);
	y = (posY + 1);
	if (posX == 0)
		x = filas - 1;
	if (posY == (columnas - 1))
		y = 0;
	if (bloques[x*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

		
	}

	//DERECHA  % columnas
	y = (posY + 1);
	if (posY == (columnas - 1))
		y = 0;
	if (bloques[posX*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

		
	}

	//ABAJO-DERECHA
	x = (posX + 1);
	y = (posY + 1);
	if (posX == (filas - 1))
		x = 0;
	if (posY == (columnas - 1))
		y = 0;
	if (bloques[x*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

		
	}

	//ABAJO % filas
	x = (posX + 1);
	if (posX == (filas - 1))
		x = 0;
	if (bloques[x*filas + posY].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

		
	}

	//ABAJO-IZQU
	x = (posX + 1);
	y = (posY - 1);
	if (posX == (filas - 1))
		x = 0;
	if (posY == 0)
		y = columnas - 1;
	if (bloques[x*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;

	}

	//IZQU % columnas
	y = (posY - 1);
	if (posY == 0)
		y = columnas - 1;
	
	if (bloques[posX*filas + y].color == bloques[posX*filas + posY].color){
		bloquesAdyacentes = bloquesAdyacentes + 1;
		
	}
	
	return bloquesAdyacentes;
}		




__device__ Block obtenerBloqueSiguiente(Block* dev_bloques, Block bloqueActual, int tx, int ty, int filas, int columnas, int colorInit, bool bomba) {
	int bloquesProximos = 0;
	if(bomba){
		
	}
	bloquesProximos = bloquesIgualesAdyacentes(dev_bloques, filas, columnas, tx, ty);    
	if (bloquesProximos >= 2 && bloqueActual.color == colorInit) {	
		bloqueActual.color = 0;
	}
	
	return bloqueActual;  

}



//MUEVE CEROS HACIA ARRIBA
int colocarArriba(Block *bloques, int i, int filas, int columnas){
	bool limite=false;
	int x=i;                                       //No queremos perder la posicion del primer bloque por eso la guardamos en x
	
	while(limite==false && bloques[x].color==0){  //Es para saber la posicion con el que hay que cambiar 
		
		if((x-columnas)>=0) x=x-columnas;    //cambiamos las filas hasta encontrar la posicion deseada
		else limite=true;                   //Limite es control de desbordo
	}
	
	bloques[i].color=bloques[x].color;
	bloques[x].color=0;
	
	return 0;
}


////MUEVE CEROS HACIA DERECHA
int colocarDerecha(Block *bloques, int i, int filas, int columnas){
	
	bool limite=false;
	int x=i;                                         //No queremos perder la posicion del primer bloque por eso la guardamos en x
	 
	while(limite==false && bloques[x].color==0){    //Es para saber la posicion con el que hay que cambiar el 0
		
		if(x==((filas*columnas)-1)) limite=true;    //Limite es control de desbordo
		else x++;                                  //subimos posicion
	}
	
	//Intercambiamos las columnas
	while(i>0 && x>0){//limite
		bloques[i].color=bloques[x].color;
		bloques[x].color=0;
		i=i-columnas;
		x=x-filas;
	}

	return 0;
}

//MOVER BLOQUES
int moveBlocks(Block *bloques, int filas, int columnas){

	int j=0;
		
	for(j=(((filas*columnas)-1));j>=0;j--){
		if(bloques[j].color==0){
			
			colocarArriba(bloques,j,filas,columnas);
			
		}
	}
	   
	for(j=(((filas*columnas)-1)-columnas);j<((filas*columnas)-1);j++){
		
		if(bloques[j].color==0){                                      //comprobamos la fila de mas abajo 
			colocarDerecha(bloques,j,filas,columnas);
		}
	}

	return 0;
}


Block *explotarBomba(Block *bloques, int posX, int posY, int filas, int columnas){

	
	////Explotar centro-arriba   

	if (posX!=0){
		if(bloques[(posX-1)*columnas+posY].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX-1)*columnas+posY].color=0; 
		}
	}

	if ((posX-2)>=0){
		if(bloques[(posX-2)*columnas+posY].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX-2)*columnas+posY].color=0;
		}
	}
	

	//Explotar centro-abajo 

	if (posX!=filas-1){
		if(bloques[(posX+1)*columnas+posY].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX+1)*columnas+posY].color=0;
		}
	}

	if (posX+2<=filas-1){
		if(bloques[(posX+2)*columnas+posY].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX+2)*columnas+posY].color=0;
		}
	}
	
	//Explotar derecha
	if (posY!=columnas-1){
	
		if(bloques[posX*columnas+(posY+1)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX*columnas)+(posY+1)].color=0;
		}
	}

	if (posY+2 < columnas-1){
	
		if(bloques[posX*columnas+(posY+2)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
			
		}
		else{
			bloques[(posX*columnas)+(posY+2)].color=0;
			
		}
	}


	//Explotar izquierda
	if (posY!=0){
		
		if(bloques[posX*columnas+(posY-1)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX*columnas)+(posY-1)].color=0;
		}
	}

	if (posY-2 > 0){
		
		if(bloques[posX*columnas+(posY-2)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
			
		}
		else{
			bloques[(posX*columnas)+(posY-2)].color=0;
			
		}
	}


	//Explotar derecha-abajo

	if (posX!=(filas-1) && posY!=(columnas-1)){
		
		if(bloques[(posX+1)*columnas+(posY+1)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX+1)*columnas+(posY+1)].color=0; 
		}
	}

	//Explotar derecha-arriba

	if (posX!=0 && posY!=(columnas-1)){
		
		if(bloques[(posX-1)*columnas+(posY+1)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX-1)*columnas+(posY+1)].color=0; 
		}
	}

	//Explotar izquierda-arriba
	
	if (posX!=0 && posY!=0){
		
		if(bloques[(posX-1)*columnas+(posY-1)].color==9){
			bloques[(posX-1)*columnas+(posY-1)].color=0;
		}
		else{
			bloques[(posX-1)*columnas+(posY-1)].color=0; 
		}
	}

	//Explotar izquierda-abajo

	if (posX!=(filas-1) && posY!=0){
		
		if(bloques[(posX+1)*columnas+(posY-1)].color==9){
			bloques[(posX+1)*columnas+(posY-1)].color=0; 
		}
		else{
			bloques[(posX+1)*columnas+(posY-1)].color=0; 
		}
	
	bloques[posX*filas + posY].color = 0;
	
	}
	return bloques;
	
}



int main(int argc, char *argv[]) {
	bool modoAutomatico;

	HANDLE hConsole;
	hConsole = GetStdHandle(STD_OUTPUT_HANDLE);

	int filas;
	int columnas;

	//string modo = "-m";
	string modo = "-m";
	cout<<"Introduce el n�mero de filas y columnas:_";
		int x=0, y=0;
		cout<<"\nfilas: ";
		cin>>filas;
		cout<<"\ncolumnas: ";
		cin>>columnas;


	cout<< "Filas y columnas cargadas, pulse enter para continuar";

	getchar();

	/* Establece el modo de ejecuci�n en funcion de los datos introducidos */
	if (modo.compare("-a") == 0){
		modoAutomatico = true;
	}
	else if (modo.compare("-m") == 0)
		modoAutomatico = false;
	else {
		cout << "\nEl modo de ejecucion introducido es incorrecto";
		exit(1);
	}


	//Creamos los arrays de bloques
	Block* bloques = new Block[filas * columnas];
	Block* bloquesSiguientes = new Block[filas * columnas];
	Block *dev_bloques;
	Block *dev_bloquesSiguientes;

	int numAleatorio;


	for (int i =0; i < filas; i++){  //llenamos el array de nums aleatorios
		for (int j = 0; j < columnas; j++) {
			numAleatorio = rand() % nColores;
			bloques[(i*filas) + j] = Block(i, j);
			bloques[(i*filas) + j].color = numAleatorio + 1;
			
		}
	}

	while (true) {

		
		cout << "\n\n\n";

		for (int i = 0; i<filas; i++) {
			for (int j = 0; j<columnas; j++)
				bloques[i*columnas + j].printBloque(hConsole); 
			cout << "\n";
		}
		cout << "\n\n\n";
		SetConsoleTextAttribute(hConsole, 15);

		int x=0, y=0;
		cout<<"Introduce la posicion del bloque que quieras explotar o 99 para salir";
			
		cout<<"\nfila: ";
		cin>>x;

		if(x == 99)
			exit(0);
			
		cout<<"columna: ";
		cin>>y;
		
		

		if(bloques[x*columnas + y].color == 9){ //se trata de una bomba, explotamos la cruz de bloques
			explotarBomba(bloques, x, y, filas, columnas);
			
		}
		else{

		int size = filas*columnas*sizeof(Block);

		//Reservamos memoria
		(hipMalloc((void**)&dev_bloques, size));
		(hipMalloc((void**)&dev_bloquesSiguientes, size));

		//Transferencia de datos de la memoria CPU al Device
		(hipMemcpy(dev_bloques, bloques, size, hipMemcpyHostToDevice));

		//Dimensiones de cada bloque
		dim3 dimBlock(filas, columnas);

		//Llamada del m�todo del Kernel. Tendremos 2 bloques/grid y dentro de �ste, tantos hilos como posiciones tenga el tablero
		bloquesKernel << <1, dimBlock >> >(dev_bloques, dev_bloquesSiguientes, filas, columnas, x, y);

		//Transferir la variable del dispositivo al host
		(hipMemcpy(bloquesSiguientes, dev_bloquesSiguientes, size, hipMemcpyDeviceToHost));

		//Liberar la memoria del dipositivo
		hipFree(dev_bloquesSiguientes);
		hipFree(dev_bloques);


		if (!modoAutomatico)
			system("pause");
		else
			Sleep(2000);
			system("pause");
		bloques = bloquesSiguientes;
		
		}

		moveBlocks(bloques, filas, columnas);

	}


	free(bloques);
	free(bloquesSiguientes);
}


