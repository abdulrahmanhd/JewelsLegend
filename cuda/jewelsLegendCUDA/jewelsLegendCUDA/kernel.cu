#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <assert.h>
#include <cmath>
#include <Windows.h>


//Poniendo este DEFINE evitamos un error en el que falta la definici�n de HANDLE_ERROR
#define HANDLE_ERROR
#define COLORES 2
#define ID_BOMBA 8

enum posicion {arriba, abajo, derecha, izquierda };


//Funci�n que devuelve un error si las dimensiones de la martiz son demasiado grandes para la gr�fica
hipError_t comprobarPropiedades(int filas, int columnas) {
	hipDeviceProp_t prop;
	hipError_t cudaStatus = hipSuccess;
	int count;
	long globalMem;
	int sharedMem;
	HANDLE_ERROR(hipGetDeviceCount(&count));

	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		globalMem = prop.totalGlobalMem;
		sharedMem = prop.sharedMemPerBlock;

		printf("La matriz solicitada ocupa %lu\n", filas*columnas * sizeof(int));
		printf("La memoria global es de %lu\n", globalMem);
		printf("La memoria compartida es de %d\n", sharedMem);

		if ((filas*columnas * sizeof(int)) >= globalMem) {

			fprintf(stderr, "La matriz solicitada ocupa %lu y excede la capacidad de memoria global de tu tarjeta gr�fica que es %lu \n",
				filas*columnas * sizeof(int), globalMem);
			goto Error;
		}


		if ((filas*columnas * sizeof(int)) >= sharedMem) {

			fprintf(stderr, "La matriz solicitada ocupa %lu y excede la capacidad de memoria compartida de tu tarjeta gr�fica que es %lu \n",
				filas*columnas * sizeof(int), sharedMem);
			goto Error;
		}


	}

Error:
	return cudaStatus;

}

//Funcion que comprueba los estaDentro de la matriz, ya que tratamos con un array
__device__ bool estaDentro(int x, int y, int filas, int columnas) {

	return !(x >= filas || x < 0 || y >= columnas || y < 0);

}

//funcion que comprueba las posiciones iguales del array,
__device__ int comprobarIgualesPos(int *tablero, int posX, int posY, posicion pos, int tamFilas,int tamColumnas) {
	int cont = 0;
	switch (pos)
	{
	case derecha:
		if (posY + 1 < tamColumnas && tablero[(posX*tamFilas) + posY] == tablero[(posX * tamFilas) + posY + 1]) { // comprobamos derecha 
			cont += 1 + comprobarIgualesPos(tablero, posX, posY + 1, derecha, tamFilas,tamColumnas);
		}
		break;
	case izquierda:
		if (posY - 1 >= 0 && tablero[(posX*tamFilas) + posY] == tablero[(posX * tamFilas) + posY - 1]) { //comprobamos izquierda
			cont += 1 + comprobarIgualesPos(tablero, posX, posY - 1, izquierda, tamFilas,tamColumnas);
		}
		break;
	case abajo:
		if (posX + 1 < tamFilas && tablero[(posX*tamFilas) + posY] == tablero[((posX + 1) * tamFilas) + posY]) { //comprobamos abajo
			cont += 1 + comprobarIgualesPos(tablero, posX + 1, posY, abajo, tamFilas, tamColumnas);
		}
		break;
	case arriba:
		if (posX - 1 >= 0 && tablero[(posX*tamFilas) + posY] == tablero[((posX - 1) * tamFilas) + posY]) { //comprobamos arriba
			cont += 1 + comprobarIgualesPos(tablero, posX - 1, posY, arriba, tamFilas, tamColumnas);
		}
		break;
	default:
		break;
	}

	return cont;

}

//Funcion que elimina una celda
__device__ void eliminar(int* dev_tablero, int fila, int columna, int tamFilas, int* dev_contadorEliminados) {

	// Si el valor examinado es distinto de 0, se suma uno al contador de eliminador
	if (dev_tablero[fila * tamFilas + columna] != 0)
		dev_contadorEliminados[0] = dev_contadorEliminados[0] + 1;

	//El valor se pone a 0
	dev_tablero[fila * tamFilas + columna] = 0;


}
// Funcion que determina si alineacion 1 es mayoor que 2 o viceversa
__device__ bool comprobarMayor(int sameVertical1, int sameHorizon1, int sameVertical2, int sameHorizon2) {
	bool mayor1 = true;
	if (((sameVertical1 >= sameVertical2) && (sameVertical1 >= sameHorizon2)) || ((sameHorizon1 >= sameVertical2) && (sameHorizon1 >= sameHorizon2)))	mayor1 = true;
	else mayor1 = false;
	return mayor1;
}


// Funcion que elimina con un unico bloque
__device__ void comprobarCadena(int* dev_tablero, int fila1, int columna1, int fila2, int columna2,int tamFilas, int tamColumnas, int* dev_contadorEliminados) {
	
	

	//Valores de los indices
	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y
	//eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados);
	
	int sameVertical1 = comprobarIgualesPos(dev_tablero, fila1, columna1, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila1, columna1, arriba, tamFilas, tamColumnas);
	int sameHorizon1 = comprobarIgualesPos(dev_tablero, fila1, columna1, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila1, columna1, izquierda, tamFilas, tamColumnas);
	int sameVertical2 = comprobarIgualesPos(dev_tablero, fila2, columna2, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, arriba, tamFilas, tamColumnas);
	int sameHorizon2 = comprobarIgualesPos(dev_tablero, fila2, columna2, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, izquierda, tamFilas, tamColumnas);
	
	

	if (comprobarMayor(sameVertical1,sameHorizon1,sameVertical2,sameHorizon2)) { //comprobamos cual de las posiciones cambiadas explota mas
	//Ahora comprobamos que el hilo sea el de la posicion que queremos
		if (i == fila1 && j == columna1) {
			if (sameHorizon1 > sameVertical1) {
				int jAux = j;
				while (dev_tablero[i*tamFilas + j] == dev_tablero[i*tamFilas + (jAux + 1)] && jAux + 1 < tamColumnas) { //eliminamos igual por derecha
					jAux++;
					eliminar(dev_tablero, i, jAux, tamFilas, dev_contadorEliminados);
					
				}
				jAux = j;
				while (dev_tablero[i*tamFilas + j] == dev_tablero[i*tamFilas + (jAux - 1)] && jAux - 1 >= 0) {//eliminamos igual por izquierda
					jAux--;
					eliminar(dev_tablero, i, jAux , tamFilas, dev_contadorEliminados);
				}
				eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados); //eliminamos la posicion del hilo
			}
			else { //if(mayor1){
				if (comprobarMayor(sameVertical1, sameHorizon1, sameVertical2, sameHorizon2)) {
					int iAux = i;
					while (dev_tablero[i*tamFilas + j] == dev_tablero[(iAux + 1)*tamFilas + j] && iAux + 1 < tamFilas) { //eliminamos igual por arriba
						iAux++;
						eliminar(dev_tablero, iAux, j, tamFilas, dev_contadorEliminados);

					}
					iAux = i;
					while (dev_tablero[i*tamFilas + j] == dev_tablero[(iAux - 1)*tamFilas + j] && iAux - 1 >= 0) {//eliminamos igual por abajo
						iAux--;
						eliminar(dev_tablero, iAux, j, tamFilas, dev_contadorEliminados);
					}
					eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados); //eliminamos la posicion del hilo
				}
			}
		}
	}
	else {
		if (i == fila2 && j == columna2) {
			if (sameHorizon2 > sameVertical2) {
				int jAux = j;
				while (dev_tablero[i*tamFilas + j] == dev_tablero[i*tamFilas + (jAux + 1)] && jAux + 1 < tamColumnas) { //eliminamos igual por derecha
					jAux++;
					eliminar(dev_tablero, i, jAux, tamFilas, dev_contadorEliminados);
				}
				jAux = j;
				while (dev_tablero[i*tamFilas + j] == dev_tablero[i*tamFilas + (jAux - 1)] && jAux - 1 >= 0) {//eliminamos igual por izquierda
					jAux--;
					eliminar(dev_tablero, i, jAux, tamFilas, dev_contadorEliminados);
				}
				eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados); //eliminamos la posicion
			}
			else{// if(mayor2) {
				if (!comprobarMayor(sameVertical1, sameHorizon1, sameVertical2, sameHorizon2)) {
					int iAux = i;
					while (dev_tablero[i*tamFilas + j] == dev_tablero[(iAux + 1) * tamFilas + j] && iAux + 1 < tamFilas) { //eliminamos igual por abajo
						iAux++;
						eliminar(dev_tablero, iAux, j, tamFilas, dev_contadorEliminados);

					}
					iAux = i;
					while (dev_tablero[i * tamFilas + j] == dev_tablero[(iAux - 1) * tamFilas + j] && iAux - 1 >= 0) {//eliminamos igual por arriba
						iAux--;
						eliminar(dev_tablero, iAux, j, tamFilas, dev_contadorEliminados);
					}
					eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados); //eliminamos la posicion del hilo
				}
			}
			
		}
	}
}



// Metodo que sube los 0 que se encuentran en el tablero hacia la parte mas alta del mismo
__device__ void reestructuracionArribaAbajo(int* dev_tablero, int filas, int columnas) {

	
	int celdax = blockIdx.y* blockDim.y + threadIdx.y;		//Indice de la x
	int celday = blockIdx.x* blockDim.x + threadIdx.x;		//Indice de la y

	if(dev_tablero[celdax*filas+celday] == 0){
		int celdaxAux = celdax;
		while (celdaxAux-1 >= 0) {
			if (dev_tablero[(celdaxAux - 1)*filas + celday] != 0) {
				int colorAux = dev_tablero[(celdaxAux - 1)*filas + celday];
				dev_tablero[(celdaxAux - 1)*filas + celday] = 0;
				dev_tablero[celdaxAux*filas+celday] = colorAux;
			}
			celdaxAux--;
			
		}
	}

	/*int nombre = celdax * columnas + celday;	//Valor del elemento en el array
	int actual = nombre;
	int count = 0;
	int comprobador = 0;
	int size = (filas*columnas); //Tama�o de la matriz
	// Se comprueba que esa celda no es 0 para compararla con los elementos que tiene por debajo
	if (dev_tablero[actual] != 0) {

		actual += columnas;

		// Se comprueban cuantos 0 hay por debajo de la celda. Este numero se guardara en la variable count
		while (actual < size) {

			if (dev_tablero[actual] == 0) {
				count++;
			}

			actual += columnas;

		}

		//Cambio de valor de la celda que se comprueba con la celda de las posiciones que tiene que descender
		if (count > 0) {
			dev_tablero[nombre + (count * columnas)] = dev_tablero[nombre];
		}

		actual = nombre - filas;

		//Comprobacion de cuantos 0 por encima tiene la celda que se comprueba
		while (actual > 0) {
			if (dev_tablero[actual] == 0) {
				comprobador++;
			}
			actual -= filas;
		}

		//Poner a 0 el valor de la celda que se cambia si su fila menos el numero de 0 que tiene por encima
		// es menor o igual que el numero de ceros que tiene por debajo
		if (celdax - comprobador < count) {
			dev_tablero[nombre] = 0;
		}

	}*/

}

__device__ void reestructuracionIzquierdaDerecha(int* dev_tablero, int filas, int columnas, int fila, int columna) {

	int size = (filas*columnas);						//Tama�o de la matriz
	int x = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int y = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y

	int comprobador = 0; // Numero de elementos que no son 0 por encima de la celda visitada que contiene un 0
	int actual;
	int auxiliar;

	//La celda eliminada es quien reestructura la matriz
	if (x == fila && y == columna) {

		//Por cada celda se comprueba si es 0
		for (int i = size; i >= 0; i--)
		{
			//Si es 0, el 0 se desplaza a la derecha tantas veces como numeros diferentes por la derecha tenga
			if (dev_tablero[i] == 0) {

				actual = i + 1;

				while (actual % columnas != 0) {
					if (dev_tablero[actual] != 0) {
						comprobador++;
					}
					actual += 1;
				}

			}

			actual = i;

			//Intercambio
			for (int k = 0; k < comprobador; k++) {

				auxiliar = dev_tablero[actual + 1];
				dev_tablero[actual + 1] = dev_tablero[actual];
				dev_tablero[actual] = auxiliar;

				actual += 1;

			}

			comprobador = 0;

		}

	}

}

__global__ void jugarKernel(int* dev_tablero, int fila1, int columna1, int fila2, int columna2,int tamFila, int tamColumnas, int* dev_contadorEliminados) {

	comprobarCadena(dev_tablero, fila1, columna1, fila2, columna2,tamFila,tamColumnas, dev_contadorEliminados);

	__syncthreads();

	//reestructuracionArribaAbajo(dev_tablero, tamFila, tamColumnas);

	__syncthreads();

	//reestructuracionIzquierdaDerecha(dev_tablero, filas, columnas, fila, columna);

	//__syncthreads();

}

__device__ bool adyacentes(int fila1, int columna1, int fila2, int columna2) {
	bool ady = false;

	if (fila1 == fila2 + 1 || fila1 == fila2 - 1 || (fila1 == fila2 && columna1 != columna2)) {
		if (columna1 == columna2 + 1 || columna1 == columna2 - 1 || (columna1 == columna2 && fila1 != fila2)) {
			if ((fila1 == fila2 - 1 && columna1 == columna2 - 1) || (fila1 == fila2 + 1 && columna1 == columna2 - 1) || (fila1 == fila2 - 1 && columna1 == columna2 + 1) || (fila1 == fila2 + 1 && columna1 == columna2 + 1)) {
				ady = false; //Si el movimiento es en diagonal no sera valido
			}
			else { ady = true; }
		}
	}
	return ady;
}

__device__ bool explotChange(int* dev_tablero, int filas1, int columnas1, int fila2, int columna2,int tamFilas,int tamColumnas) {
	bool explotan = false;

	//HAcemos el intercambio en la matriz para comprobar si se puede explotar
	int colorAux1 = dev_tablero[(filas1*tamFilas) + columnas1];
	int colorAux2 = dev_tablero[(fila2*tamFilas) + columna2];
	dev_tablero[(filas1*tamFilas) + columnas1] = colorAux2;
	dev_tablero[(fila2*tamFilas) + columna2] = colorAux1;

	int sameVertical1 = comprobarIgualesPos(dev_tablero, filas1, columnas1, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, filas1, columnas1, arriba, tamFilas, tamColumnas);
	int sameHorizon1 = comprobarIgualesPos(dev_tablero, filas1, columnas1, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, filas1, columnas1, izquierda, tamFilas, tamColumnas);
	int sameVertical2 = comprobarIgualesPos(dev_tablero, fila2, columna2, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, arriba, tamFilas, tamColumnas);
	int sameHorizon2 = comprobarIgualesPos(dev_tablero, fila2, columna2, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, izquierda, tamFilas, tamColumnas);

	if (sameVertical1 >= 2 || sameHorizon1 >= 2) { //Comprobamos que en cualquiera de las posiciones haya bombas que puedan explotar
		explotan = true;
	}
	else if (sameVertical2 >= 2 || sameHorizon2 >= 2) {
		explotan = true;
	}

	// deshacemos los cambios en la matriz
	dev_tablero[(filas1*tamFilas) + columnas1] = colorAux1;
	dev_tablero[(fila2*tamFilas) + columna2] = colorAux2;

	return explotan;
}


__global__ void probeMovPosi(int* dev_tablero, int filas1, int columnas1, int fila2, int columna2,int tamFilas,int tamColumnas,bool* dev_mov) {
	int col = threadIdx.x;
	int fil = threadIdx.y;
	bool TrueMov;
	bool explot;
	if ((col == columnas1 && fil == filas1)) {//comprobamos que el hilo que accede a la funcion sea el que queremos cambiar(se comprueban en las funciones los dos numeros)
		TrueMov = adyacentes(filas1, columnas1, fila2, columna2);
		explot = explotChange(dev_tablero, filas1, columnas1, fila2, columna2,tamFilas,tamColumnas);
		if (TrueMov && explot) {
			//Si ambos son true realizamos el cmbio
			int colorAux1 = dev_tablero[(filas1*tamFilas) + columnas1];
			int colorAux2 = dev_tablero[(fila2*tamFilas) + columna2];
			dev_tablero[(filas1*tamFilas) + columnas1] = colorAux2;
			dev_tablero[(fila2*tamFilas) + columna2] = colorAux1;
			*dev_mov = true;
		}
		else { printf("MOVIMIENTO ERRONEO, Las posiciones no son adyacentes o no explotan\n"); }
	}
}

hipError_t jugar(int* tablero, int tamFilas, int tamColumnas, int* contadorEliminados, char m) {

	hipError_t cudaStatus;
	int fila1 = 0, columna1 = 0, fila2 = 0, columna2 = 0;
	bool* mov = false, *dev_mov;
	int* dev_tablero;
	int *dev_contadorEliminados;

	dim3 blocks(1);
	dim3 threads(tamFilas, tamColumnas);
	//Asignamos objeto a memoria global con cudamalloc
	cudaStatus = hipMalloc((void**)&dev_tablero, tamFilas*tamColumnas * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_contadorEliminados, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_mov, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Pasamos parametros a la parte del device
	cudaStatus = hipMemcpy(dev_tablero, tablero, tamFilas*tamColumnas * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_tablero failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_contadorEliminados, contadorEliminados, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_contadorEliminados failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mov, &mov, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_mov failed!");
		goto Error;
	}
	//Modo manual
	if (m == 'm') {
		while (!mov) {
			printf("Introduzca la fila del primer diamante: ");
			scanf_s("%d", &fila1);
			printf("Introduzca la columna del primer diamante: ");
			scanf_s("%d", &columna1);
			printf("Introduzca la fila del segundo diamante: ");
			scanf_s("%d", &fila2);
			printf("Introduzca la columna del segundo diamante: ");
			scanf_s("%d", &columna2);
			
			//bool canMove = hasMoreMovements(tablero);
			probeMovPosi<< <blocks, threads>> >(dev_tablero, fila1, columna1, fila2, columna2, tamFilas, tamColumnas,dev_mov);

			cudaStatus = hipMemcpy(&mov, dev_mov, sizeof(bool), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy device to host dev_mov failed!");
				goto Error;
			}

		}

	}
	else {
		/*AUTOMATICO
		while (tablero[fila *columnas + columna] == 0) {
			fila = 0 + (rand() % filas);
			columna = 0 + (rand() % columnas);
		}

		printf("Numero de fila: %d\n", fila);
		printf("Numero de columna: %d\n", columna);*/
	}

	jugarKernel<< <blocks, threads >> >(dev_tablero, fila1, columna1, fila2, columna2, tamFilas, tamColumnas, dev_contadorEliminados);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "jugarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(tablero, dev_tablero, tamFilas*tamColumnas * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy tablero failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(contadorEliminados, dev_contadorEliminados, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy contadorEliminados failed!");
		goto Error;
	}


Error:
	hipFree(dev_tablero);
	hipFree(dev_contadorEliminados);
	hipFree(dev_mov);

	return cudaStatus;

}

int* generaTablero(int filas, int columnas, int bombas);
void imprimeTablero(int* tablero, int filas, int columnas);
char pedirModoEjecucion();
int pedirFilasTablero();
int pedirColumnasTablero();
char pedirDificultad();
void prop();

int main() {
	//Declaracion de variables para la ejecucion del programa

	hipError_t cudaStatus;
	//prop();
	int tamFilas; //Filas que tendra el tablero del programa
	int tamColumnas; //Columnas que tendra el tablero del programa
	char modo; //Modo de ejecucion del programa
	int contadorEliminados = 0;
	char dificultad;
	int* tablero;
	int nColores;

	modo = pedirModoEjecucion();
	dificultad = pedirDificultad();
	tamFilas = pedirFilasTablero();
	tamColumnas = pedirColumnasTablero();

	if (dificultad == 'F') {
		nColores = 4;
	}
	else if (dificultad == 'M') {
		nColores = 6;
	}
	else {
		nColores = 8;
	}

	printf("\nLos datos introducidos por el usuario son: -%c %c %d %d\n", modo,dificultad, tamFilas, tamColumnas);

	cudaStatus = comprobarPropiedades(tamFilas, tamColumnas);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	tablero = generaTablero(tamFilas, tamColumnas, nColores);


	do {
		imprimeTablero(tablero, tamFilas, tamColumnas);

		if (modo == 'm') {
			cudaStatus = jugar(tablero, tamFilas, tamColumnas, &contadorEliminados, 'm');
		}
		else
		{
			cudaStatus = jugar(tablero, tamFilas, tamColumnas, &contadorEliminados, 'a');
		}

		imprimeTablero(tablero, tamFilas, tamColumnas);

		printf("Contador  = %d\n ", contadorEliminados);

	} while ((cudaStatus == 0) && (contadorEliminados < 100));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Algo ha fallado!");
		goto Error;
	}

	printf(" - - - - - - JUEGO TERMINADO - - - - - - - ");

Error:

	getchar();
	getchar();


	return cudaStatus;
}

//Procedimiento que imprime una matriz de filas * columnas de enteros indicando fila y columna 
void imprimeTablero(int* tablero, int filas, int columnas) {

	printf("\n     ------TABLERO------\n\n      ");

	//Imprime el numero de columna
	for (int i = 0; i < columnas; i++) {

		if (i > 99)printf("%d ", i);
		else if (i > 9) printf("%d  ", i);
		else printf("%d   ", i);

	}

	printf("\n");
	for (int i = 0; i < filas*columnas; i++) {

		//Imprime el numero de fila
		if (i % columnas == 0) {

			printf("\n");

			if ((i / columnas) > 99)printf("%d | ", i / columnas);
			else if ((i / columnas) > 9) printf("%d  | ", i / columnas);
			else printf("%d   | ", i / columnas);


		}
		//Seg�n el valor en la posici�n i del tablero se imprime de un color u otro
		switch (tablero[i]) {
		case 0: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 0);
			break;
		case 1: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1);
			break;
		case 2: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 2);
			break;
		case 3: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12);
			break;
		case 4: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13);
			break;
		case 5: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14);
			break;
		case 6: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			break;
		case 7: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 3);
			break;
		case 8: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 10);
			break;
		case 9: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 11);
			break;
		default:;
		}
		printf("%d", tablero[i]);

		//Se imprime de nuevo en blanco
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		printf(" | ");

	}


	printf("\n\n");

}


/*
Procedimiento que genera un tablero de size x size relleno con numeros del 1 al 2
y con bombas representadas con el n�mero 3
*/
int* generaTablero(int filas, int columnas, int nColores) {

	//Reserva de memoria para el numero de columnas
	int* tablero = (int*)malloc(filas*columnas * sizeof(int));

	for (int i = 0; i < filas*columnas; i++)
	{

			tablero[i] = 1 + (rand() % nColores);

	}
	return tablero;

}

char pedirDificultad() {
	char dificultad=' ';
	getchar();
	while (dificultad != 'F' && dificultad != 'M' && dificultad != 'D') {
		printf("Que dificultad desea para el juego? Facil(F), Medio(M), Dificil(D)\n");
		fflush(stdin);
		scanf("%c", &dificultad);
		if (dificultad != 'F' && dificultad != 'M' && dificultad != 'D' ) {
			printf("Usted ha introducido una dificutad no existente: -%c.\n", dificultad);
			printf("Por favor, introduzca uno de las dificultades que se le presentan a continuacion.\n\n");
		}

	};
	
	return dificultad;

}

//Metodo que solicita al usuario el modo de ejecucion del programa
char pedirModoEjecucion() {
	char modo;
	do {
		printf("Existen 2 modos de ejecucion para Jewels Leyend:\n\n");
		printf("- Automatica(a): el programa pulsara aleatoriamente las teclas del tablero\n");
		printf("- Manual(m): el programa esperara a que el usuario pulse las teclas del tablero\n");
		printf("Introduce el modo de ejecucion del programa: "); 
		fflush(stdin);
		scanf("%c", &modo);
		if (modo != 'a' && modo != 'm') {
			printf("Usted ha introducido un modo de ejecucion no existente: %c.\n", modo);
			printf("Por favor, introduzca uno de los modos que se le presentan a continuacion.\n\n");
		}
	} while (modo != 'a' && modo != 'm');
	return modo;
}

//Metodo que solicita al usuario el numero de filas que tendra el tablero
int pedirFilasTablero() {
	int filas;

	do {
		printf("\nIntroduzca las filas que tendra el tablero: ");
		fflush(stdin);
		scanf_s("%d", &filas);
		if (filas < 1 || filas > 2147483647) {
			printf("Introduzca un numero de filas correcto\n");
		}
	} while (filas < 1 || filas > 2147483647); //El numero de filas tiene que ser un numero entero positivo
	return filas;
}

//Metodo que solicita al usuario el numero de columnas que tendra el tablero
int pedirColumnasTablero() {
	int columnas;
	do {
		printf("\nIntroduzca las columnas que tendra el tablero: ");
		fflush(stdin);
		scanf_s("%d", &columnas);
		if (columnas < 1 || columnas > 2147483647) {
			printf("Introduzca un numero de columnas correcto\n");
		}
	} while (columnas < 1 || columnas > 2147483647); //El numero de filas tiene que ser un numero entero positivo
	return columnas;
}
/*
bool hasMoreMovements(int *tablero) {
	bool expl = false;
	int posX = 0;
	while (posX < filas && !expl) {
		for (int posY = 0; posY < columnas; posY++) {
			if (posX + 1 < filas && explotan(tablero, posX, posY, posX + 1, posY, true)) { // Abajo
				expl = true;
			}
			else if (posY + 1 < columnas && explotan(tablero, posX, posY, posX, posY + 1, true)) { //Derecha
				expl = true;
			}
			else if (posY - 1 >= 0 && explotan(tablero, posX, posY, posX, posY - 1, true)) { //Izquierda
				expl = true;
			}
			else if (posX - 1 >= 0 && explotan(tablero, posX, posY, posX - 1, posY, true)) {//Arriba
				expl = true;
			}
		}
		posX++;
	}

	return expl;
}*/


void prop() {

	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execition timeout : ");



		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %lu\n", prop.totalGlobalMem);
		printf("Total constant Mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n",
			prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n",
			prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);

		printf("\n");
	}

}