#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <assert.h>
#include <cmath>
#include <Windows.h>
#include <string.h>
#include <time.h>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <iostream>
# include <stdio.h>
# include <conio.h>


//Poniendo este DEFINE evitamos un error en el que falta la definici�n de HANDLE_ERROR
#define HANDLE_ERROR


int* generaTablero(int filas, int columnas, int bombas);
void imprimeTablero(int* tablero, int filas, int columnas);
char pedirModoEjecucion();
int pedirFilasTablero();
int pedirColumnasTablero();
char pedirDificultad();
void prop();
int* rellenarTablero(int* tablero, int tamFilas, int tamColumnas, int nColores);
enum posicion {arriba, abajo, derecha, izquierda,ArribaIzquierda,ArribaDerecha, AbajoIzquierda,AbajoDerecha};
void guardarPartida(int* tablero, int filas, int columnas, int dificultad);
FILE *doc;
FILE *leer;

//Funci�n que devuelve un error si las dimensiones de la martiz son demasiado grandes para la gr�fica
hipError_t comprobarPropiedades(int filas, int columnas) {
	hipDeviceProp_t prop;
	hipError_t cudaStatus = hipSuccess;
	int count;
	long globalMem;
	int sharedMem;
	HANDLE_ERROR(hipGetDeviceCount(&count));

	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		globalMem = prop.totalGlobalMem;
		sharedMem = prop.sharedMemPerBlock;

		printf("La matriz solicitada ocupa %lu\n", filas*columnas * sizeof(int));
		printf("La memoria global es de %lu\n", globalMem);
		printf("La memoria compartida es de %d\n", sharedMem);

		if ((filas*columnas * sizeof(int)) >= globalMem) {

			fprintf(stderr, "La matriz solicitada ocupa %lu y excede la capacidad de memoria global de tu tarjeta gráfica que es %lu \n",
				filas*columnas * sizeof(int), globalMem);
			goto Error;
		}


		if ((filas*columnas * sizeof(int)) >= sharedMem) {

			fprintf(stderr, "La matriz solicitada ocupa %lu y excede la capacidad de memoria compartida de tu tarjeta gráfica que es %lu \n",
				filas*columnas * sizeof(int), sharedMem);
			goto Error;
		}


	}

Error:
	return cudaStatus;

}

int comprobarIgualesArriba(int *tablero, int posX, int posY,int tamColumnas) {
	int cont = 0;
	if (posX - 1 >= 0 && tablero[(posX*tamColumnas) + posY] == tablero[((posX - 1) * tamColumnas) + posY]) {
		cont = 1 + comprobarIgualesArriba(tablero, posX - 1, posY, tamColumnas);
	}
	return cont;
}
int comprobarIgualesIzquierda(int *tablero, int posX, int posY, int tamColumnas) {
	int cont = 0;
	if (posY - 1 >= 0 && tablero[(posX*tamColumnas) + posY] == tablero[(posX * tamColumnas) + posY - 1]) {
		cont = 1 + comprobarIgualesIzquierda(tablero, posX, posY - 1, tamColumnas);
	}
	return cont;
}

int comprobarIgualesDer(int*tablero, int posX, int posY,int tamColumnas) {
	int cont = 0;
	if (posY + 1 < tamColumnas && tablero[(posX*tamColumnas) + posY] == tablero[(posX * tamColumnas) + posY + 1]) {
		cont = 1 + comprobarIgualesDer(tablero, posX, posY + 1, tamColumnas);
	}
	return cont;
}
int comprobarIgualesAbajo(int *tablero, int posX, int posY, int tamColumnas, int tamFilas) {
	int cont = 0;
	//if(posX >= filas - 1 && posY >= columnas - 1)
	if (posX + 1 < tamFilas && tablero[(posX*tamColumnas) + posY] == tablero[((posX + 1) * tamColumnas) + posY]) {
		cont = 1 + comprobarIgualesAbajo(tablero, posX + 1, posY,tamFilas, tamColumnas);
	}
	return cont;
}
bool explotan(int* dev_tablero, int fila1, int columna1, int fila2, int columna2, int tamFilas, int tamColumnas) {
	bool explotan = false;

	//HAcemos el intercambio en la matriz para comprobar si se puede explotar
	int colorAux1 = dev_tablero[(fila1*tamColumnas) + columna1];
	int colorAux2 = dev_tablero[(fila2*tamColumnas) + columna2];
	dev_tablero[(fila1*tamColumnas) + columna1] = colorAux2;
	dev_tablero[(fila2*tamColumnas) + columna2] = colorAux1;

	int sameHorizon1 = comprobarIgualesDer(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesIzquierda(dev_tablero, fila1, columna1, tamColumnas);
	int sameVertical1 = comprobarIgualesArriba(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesAbajo(dev_tablero, fila1, columna1, tamColumnas, tamFilas);

	int sameHorizon2 = comprobarIgualesDer(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesIzquierda(dev_tablero, fila1, columna1, tamColumnas);
	int sameVertical2 = comprobarIgualesArriba(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesAbajo(dev_tablero, fila1, columna1, tamColumnas, tamFilas);


	// deshacemos los cambios en la matriz
	dev_tablero[(fila1*tamColumnas) + columna1] = colorAux1;
	dev_tablero[(fila2*tamColumnas) + columna2] = colorAux2;

	if (sameVertical1 >= 2 || sameHorizon1 >= 2) { //Comprobamos que en cualquiera de las posiciones haya bombas que puedan explotar
		explotan = true;
	}
	else if (sameVertical2 >= 2 || sameHorizon2 >= 2) {
		explotan = true;
	}



	return explotan;
}
bool hasMoreMovements(int *tablero, int filas, int columnas) {
	bool expl = false;
	int posX = 0;
	while (posX < filas && !expl) {
		for (int posY = 0; posY < columnas; posY++) {
			if (posX + 1 < filas && explotan(tablero, posX, posY, posX + 1, posY, filas, columnas)) { // Abajo
				expl = true;
			}
			else if (posY + 1 < columnas && explotan(tablero, posX, posY, posX, posY + 1, filas, columnas)) { //Derecha
				expl = true;
			}
			else if (posY - 1 >= 0 && explotan(tablero, posX, posY, posX, posY - 1, filas, columnas)) { //Izquierda
				expl = true;
			}
			else if (posX - 1 >= 0 && explotan(tablero, posX, posY, posX - 1, posY, filas, columnas)) {//Arriba
				expl = true;
			}
		}
		posX++;
	}

	return expl;
}

__device__ int comprobarIgualesPos(int *tablero, int posX, int posY, posicion pos, int tamFilas, int tamColumnas) {
	int cont = 0;
	switch (pos)
	{
	case derecha:
		if (posY + 1 < tamColumnas && tablero[(posX*tamColumnas) + posY] == tablero[(posX * tamColumnas) + posY + 1]) { // comprobamos derecha 
			cont += 1 + comprobarIgualesPos(tablero, posX, posY + 1, derecha, tamFilas, tamColumnas);
		}
		break;
	case izquierda:
		if (posY - 1 >= 0 && tablero[(posX*tamColumnas) + posY] == tablero[(posX * tamColumnas) + posY - 1]) { //comprobamos izquierda
			cont += 1 + comprobarIgualesPos(tablero, posX, posY - 1, izquierda, tamFilas, tamColumnas);
		}
		break;
	case abajo:
		if (posX + 1 < tamFilas && tablero[(posX*tamColumnas) + posY] == tablero[((posX + 1) * tamColumnas) + posY]) { //comprobamos abajo
			cont += 1 + comprobarIgualesPos(tablero, posX + 1, posY, abajo, tamFilas, tamColumnas);
		}
		break;
	case arriba:
		if (posX - 1 >= 0 && tablero[(posX*tamColumnas) + posY] == tablero[((posX - 1) * tamColumnas) + posY]) { //comprobamos arriba
			cont += 1 + comprobarIgualesPos(tablero, posX - 1, posY, arriba, tamFilas, tamColumnas);
		}
		break;
	case ArribaIzquierda:
		if (posX - 1 >= 0 && posY>=0 && tablero[(posX*tamColumnas) + posY] == tablero[((posX - 1) * tamColumnas) + posY-1]) { //comprobamos arribaIzquierda
			cont += 1 + comprobarIgualesPos(tablero, posX - 1, posY-1, ArribaIzquierda, tamFilas, tamColumnas);
		}
		break;
	case ArribaDerecha:
		if (posX - 1 >= 0 && posY+1<tamColumnas && tablero[(posX*tamColumnas) + posY] == tablero[((posX - 1) * tamColumnas) + posY+1]) { //comprobamos arribaDerecha
			cont += 1 + comprobarIgualesPos(tablero, posX - 1, posY+1, ArribaDerecha, tamFilas, tamColumnas);
		}
		break;
	case AbajoIzquierda:
		if (posX + 1 < tamFilas && posY - 1 >=  0 && tablero[(posX*tamColumnas) + posY] == tablero[((posX + 1) * tamColumnas) + posY-1]) { //comprobamos abajoIzquierda
			cont += 1 + comprobarIgualesPos(tablero, posX + 1, posY-1, AbajoIzquierda, tamFilas, tamColumnas);
		}
		break;
	case AbajoDerecha:
		if (posX + 1 <tamFilas && posY+1 <tamColumnas && tablero[(posX*tamColumnas) + posY] == tablero[((posX + 1) * tamColumnas) + posY+1]) { //comprobamos abajoDerecha
			cont += 1 + comprobarIgualesPos(tablero, posX + 1, posY+1, AbajoDerecha, tamFilas, tamColumnas);
		}
		break;
	default:
		break;
	}

	return cont;

}
bool proveBig(int sameVertical1, int sameHorizon1, int sameVertical2, int sameHorizon2) {
	bool mayor1 = true;
	if (((sameVertical1 >= sameVertical2) && (sameVertical1 >= sameHorizon2)) || ((sameHorizon1 >= sameVertical2) && (sameHorizon1 >= sameHorizon2)))	mayor1 = true;
	else mayor1 = false;
	return mayor1;
}
// Funcion que determina si alineacion 1 es mayoor que 2 o viceversa
__device__ bool comprobarMayor(int sameVertical1, int sameHorizon1, int sameVertical2, int sameHorizon2) {
	bool mayor1 = true;
	if (((sameVertical1 >= sameVertical2) && (sameVertical1 >= sameHorizon2)) || ((sameHorizon1 >= sameVertical2) && (sameHorizon1 >= sameHorizon2)))	mayor1 = true;
	else mayor1 = false;
	return mayor1;
}
//Funcion que comprueba los estaDentro de la matriz, ya que tratamos con un array
__device__ bool estaDentro(int x, int y, int filas, int columnas) {

	return !(x >= filas || x < 0 || y >= columnas || y < 0);

}
//Funcion que devuelve el numero mas alto de explosiones
int autoContMov(int *dev_tablero, int fila1, int columna1, int fila2, int columna2, int tamFilas, int tamColumnas) {
	int comFilas1, comColum1, comFilas2, comColum2 = 0;

	int colorAux1 = dev_tablero[(fila1*tamColumnas) + columna1];
	int colorAux2 = dev_tablero[(fila2*tamColumnas) + columna2];

	dev_tablero[(fila1*tamColumnas) + columna1] = colorAux2;
	dev_tablero[(fila2*tamColumnas) + columna2] = colorAux1;

	comFilas1 = comprobarIgualesDer(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesIzquierda(dev_tablero, fila1, columna1,tamColumnas);
	comColum1 = comprobarIgualesArriba(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesAbajo(dev_tablero, fila1, columna1, tamColumnas, tamFilas);

	comFilas2 = comprobarIgualesDer(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesIzquierda(dev_tablero, fila1, columna1, tamColumnas);
	comColum2 = comprobarIgualesArriba(dev_tablero, fila1, columna1, tamColumnas) + comprobarIgualesAbajo(dev_tablero, fila1, columna1, tamColumnas, tamFilas);

	dev_tablero[(fila1*tamColumnas) + columna1] = colorAux1;
	dev_tablero[(fila2*tamColumnas) + columna2] = colorAux2;
	
	if (proveBig(comFilas1, comColum1, comFilas2, comColum2)) {
		if (comFilas1 > comColum1) {
			return comFilas1;
		}
		else return comColum1;
	}
	else {
		if (comFilas2 > comColum2) {
			return comFilas2;
		}
		else return comColum2;
	}
}

//funcion que comprueba las posiciones iguales del array,

//Funcion que elimina una celda
__device__ void eliminar(int* dev_tablero, int fila, int columna, int tamColumnas, int* dev_contadorEliminados) {

	// Si el valor examinado es distinto de 0, se suma uno al contador de eliminador
	if (dev_tablero[fila * tamColumnas + columna] != 0)
		dev_contadorEliminados[0] = dev_contadorEliminados[0] + 1;

	//El valor se pone a 0
	dev_tablero[fila * tamColumnas + columna] = 0;
	 

}

// Metodo que sube los 0 que se encuentran en el tablero hacia la parte mas alta del mismo
__device__ void reestructuracionArribaAbajo(int* dev_tablero, int filas, int columnas) {


	int celdax = blockIdx.y* blockDim.y + threadIdx.y;		//Indice de la x
	int celday = blockIdx.x* blockDim.x + threadIdx.x;		//Indice de la y
	int nombre = celdax * columnas + celday;	//Valor del elemento en el array
	int size = (filas*columnas); //Tamaño de la matriz
	int actual = nombre;
	int count = 0;
	int comprobador = 0;
	// Se comprueba que esa celda no es 0 para compararla con los elementos que tiene por debajo
	if (dev_tablero[actual] != 0) {

		actual += columnas;

		// Se comprueban cuantos 0 hay por debajo de la celda. Este numero se guardara en la variable count
		while (actual < size) {

			if (dev_tablero[actual] == 0) {
				count++;
			}

			actual += columnas;

		}

		//Cambio de valor de la celda que se comprueba con la celda de las posiciones que tiene que descender
		if (count > 0) {
			dev_tablero[nombre + (count * columnas)] = dev_tablero[nombre];
		}

		actual = nombre - filas;

		//Comprobacion de cuantos 0 por encima tiene la celda que se comprueba
		while (actual > 0) {
			if (dev_tablero[actual] == 0) {
				comprobador++;
			}
			actual -= filas;
		}

		//Poner a 0 el valor de la celda que se cambia si su fila menos el numero de 0 que tiene por encima
		// es menor o igual que el numero de ceros que tiene por debajo
		if (celdax - comprobador < count) {
			dev_tablero[nombre] = 0;
		}

	}

}

__device__ void reestructuracionIzquierdaDerecha(int* dev_tablero, int filas, int columnas) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y
	
	
	if (dev_tablero[i*columnas + j] == 0) {
		while (j>0) {
			dev_tablero[i*columnas + j] = dev_tablero[i*columnas + (j-1)];
			dev_tablero[i*columnas + (j-1)] = 0;
			j--;
		}
	}
}

//Elimina una fila completa
__device__ void bomba1(int* dev_tablero, int fila, int columnas) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y
	//Identificamos los diamantes por su fila
	if (i == fila)	dev_tablero[i*columnas + j] = 0;

}

//Elimina un columna completa
__device__ void bomba2(int* dev_tablero, int columna, int columnas) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y

	//Identificamos los diamantes por su columna
	if (j == columna)	dev_tablero[i*columnas + j] = 0;

}

//Mueve la matriz en fomra de cuadrados
__device__ void bomba3(int* dev_tablero, int filas , int columnas) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y
	int colorAux = 0;
	/*float fila = i%4;
	float columna = j % 4;*/
	//printf("\nEntro");
	
	if ((i == 1 && j == 1) || ((i-1) % 3 == 0 && (j-1) % 3 == 0) || ((i-1) % 3 == 0 && j==1) || (i==1 && (j-1) % 3 == 0)){
					
		if (j + 1 < columnas && i + 1 < filas) {

			//Intercambio de las puntas del cuadrado
			colorAux = dev_tablero[(i*columnas) + (j - 1)];
			dev_tablero[(i*columnas) + (j - 1)] = dev_tablero[((i + 1)*columnas) + j];
			dev_tablero[((i + 1)*columnas) + j] = dev_tablero[(i*columnas) + (j + 1)];
			dev_tablero[(i*columnas) + (j + 1)] = dev_tablero[((i - 1)*columnas) + j];
			dev_tablero[((i - 1)*columnas) + j] = colorAux;
			
			//Intercambiamos flor del cuadrado
			colorAux = dev_tablero[((i - 1)*columnas) + (j - 1)];
			dev_tablero[((i - 1)*columnas) + (j - 1)] = dev_tablero[((i + 1)*columnas) + (j - 1)];
			dev_tablero[((i + 1)*columnas) + (j - 1)] = dev_tablero[((i + 1)*columnas) + (j + 1)];
			dev_tablero[((i + 1)*columnas) + (j + 1)] = dev_tablero[((i - 1)*columnas) + (j + 1)];
			dev_tablero[((i - 1)*columnas) + (j + 1)] = colorAux;
		}
	}
	
}

//Menu de bombas
__global__ void menuBombas(int *dev_tablero, int filas, int columnas, int explota, int bomba) {
	
	switch (bomba) {
		case 91:	bomba1(dev_tablero, explota, columnas);
					reestructuracionArribaAbajo(dev_tablero, filas, columnas);
					break;
		case 92:	bomba2(dev_tablero, explota, columnas);
					reestructuracionIzquierdaDerecha(dev_tablero, filas, columnas);
					break;
		case 93:	bomba3(dev_tablero, filas, columnas);
					break;
	}
	
}

// Funcion que elimina con un unico bloque
__device__ void comprobarCadena(int* dev_tablero, int fila1, int columna1, int fila2, int columna2,int tamFilas, int tamColumnas, int* dev_contadorEliminados) {
	
	

	//Valores de los indices
	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y
	//eliminar(dev_tablero, i, j, tamFilas, dev_contadorEliminados);
	
	int sameVertical1 = comprobarIgualesPos(dev_tablero, fila1, columna1, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila1, columna1, arriba, tamFilas, tamColumnas);
	int sameHorizon1 = comprobarIgualesPos(dev_tablero, fila1, columna1, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila1, columna1, izquierda, tamFilas, tamColumnas);
	int sameVertical2 = comprobarIgualesPos(dev_tablero, fila2, columna2, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, arriba, tamFilas, tamColumnas);
	int sameHorizon2 = comprobarIgualesPos(dev_tablero, fila2, columna2, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, izquierda, tamFilas, tamColumnas);
	
	

	if (comprobarMayor(sameVertical1,sameHorizon1,sameVertical2,sameHorizon2)) { //comprobamos cual de las posiciones cambiadas explota mas
	//Ahora comprobamos que el hilo sea el de la posicion que queremos
		if (i == fila1 && j == columna1) {
			if (sameHorizon1 > sameVertical1) {
				int jAux = j;
				while (dev_tablero[i*tamColumnas + j] == dev_tablero[i*tamColumnas + (jAux + 1)] && jAux + 1 < tamColumnas) { //eliminamos igual por derecha
					jAux++;
					eliminar(dev_tablero, i, jAux, tamColumnas, dev_contadorEliminados);
					
				}
				jAux = j;
				while (dev_tablero[i*tamColumnas + j] == dev_tablero[i*tamColumnas + (jAux - 1)] && jAux - 1 >= 0) {//eliminamos igual por izquierda
					jAux--;
					eliminar(dev_tablero, i, jAux , tamColumnas, dev_contadorEliminados);
				}
				eliminar(dev_tablero, i, j, tamColumnas, dev_contadorEliminados); //eliminamos la posicion del hilo
			}
			else { //if(mayor1){
				if (comprobarMayor(sameVertical1, sameHorizon1, sameVertical2, sameHorizon2)) {
					int iAux = i;
					while (dev_tablero[i*tamColumnas + j] == dev_tablero[(iAux + 1)*tamColumnas + j] && iAux + 1 < tamFilas) { //eliminamos igual por arriba
						iAux++;
						eliminar(dev_tablero, iAux, j, tamColumnas, dev_contadorEliminados);

					}
					iAux = i;
					while (dev_tablero[i*tamColumnas + j] == dev_tablero[(iAux - 1)*tamColumnas + j] && iAux - 1 >= 0) {//eliminamos igual por abajo
						iAux--;
						eliminar(dev_tablero, iAux, j, tamColumnas, dev_contadorEliminados);
					}
					eliminar(dev_tablero, i, j, tamColumnas, dev_contadorEliminados); //eliminamos la posicion del hilo
				}
			}
		}
	}
	else {
		if (i == fila2 && j == columna2) {
			if (sameHorizon2 > sameVertical2) {
				int jAux = j;
				while (dev_tablero[i*tamColumnas + j] == dev_tablero[i*tamColumnas + (jAux + 1)] && jAux + 1 < tamColumnas) { //eliminamos igual por derecha
					jAux++;
					eliminar(dev_tablero, i, jAux, tamColumnas, dev_contadorEliminados);
				}
				jAux = j;
				while (dev_tablero[i*tamColumnas + j] == dev_tablero[i*tamColumnas + (jAux - 1)] && jAux - 1 >= 0) {//eliminamos igual por izquierda
					jAux--;
					eliminar(dev_tablero, i, jAux, tamColumnas, dev_contadorEliminados);
				}
				eliminar(dev_tablero, i, j, tamColumnas, dev_contadorEliminados); //eliminamos la posicion
			}
			else{// if(mayor2) {
				if (!comprobarMayor(sameVertical1, sameHorizon1, sameVertical2, sameHorizon2)) {
					int iAux = i;
					while (dev_tablero[i*tamColumnas + j] == dev_tablero[(iAux + 1) * tamColumnas + j] && iAux + 1 < tamFilas) { //eliminamos igual por abajo
						iAux++;
						eliminar(dev_tablero, iAux, j, tamColumnas, dev_contadorEliminados);

					}
					iAux = i;
					while (dev_tablero[i * tamColumnas + j] == dev_tablero[(iAux - 1) * tamColumnas + j] && iAux - 1 >= 0) {//eliminamos igual por arriba
						iAux--;
						eliminar(dev_tablero, iAux, j, tamColumnas, dev_contadorEliminados);
					}
					eliminar(dev_tablero, i, j, tamColumnas, dev_contadorEliminados); //eliminamos la posicion del hilo
				}
			}
			
		}
	}
}




__device__ void rellenarMatriz(int* dev_tablero, int tamFilas, int tamColumnas, int nColores) {

	//Valores de los indices
	int i = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int j = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y

	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(0, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	/* hiprand works like rand - except that it takes a state as a parameter */
	if (dev_tablero[i*tamColumnas + j] == 0) dev_tablero[i*tamColumnas + j] = hiprand(&state) % nColores;

}

__global__ void jugarKernel(int* dev_tablero, int fila1, int columna1, int fila2, int columna2,int tamFila, int tamColumnas, int* dev_contadorEliminados) {

	comprobarCadena(dev_tablero, fila1, columna1, fila2, columna2,tamFila,tamColumnas, dev_contadorEliminados);

	__syncthreads();

	reestructuracionArribaAbajo(dev_tablero, tamFila, tamColumnas);

	__syncthreads();

}

__device__ bool adyacentes(int fila1, int columna1, int fila2, int columna2) {
	bool ady = false;

	if (fila1 == fila2 + 1 || fila1 == fila2 - 1 || (fila1 == fila2 && columna1 != columna2)) {
		if (columna1 == columna2 + 1 || columna1 == columna2 - 1 || (columna1 == columna2 && fila1 != fila2)) {
			if ((fila1 == fila2 - 1 && columna1 == columna2 - 1) || (fila1 == fila2 + 1 && columna1 == columna2 - 1) || (fila1 == fila2 - 1 && columna1 == columna2 + 1) || (fila1 == fila2 + 1 && columna1 == columna2 + 1)) {
				ady = false; //Si el movimiento es en diagonal no sera valido
			}
			else { ady = true; }
		}
	}
	return ady;
}

__device__ bool explotChange(int* dev_tablero, int filas1, int columnas1, int fila2, int columna2,int tamFilas,int tamColumnas) {
	int x = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int y = blockIdx.x * blockDim.x + threadIdx.x;		//Indice de la y

	if ((x == filas1 && y == columnas1) || (x==fila2 && y == columna2)) {
		bool explotan = false;

		//HAcemos el intercambio en la matriz para comprobar si se puede explotar
		int colorAux1 = dev_tablero[(filas1*tamColumnas) + columnas1];
		int colorAux2 = dev_tablero[(fila2*tamColumnas) + columna2];
		dev_tablero[(filas1*tamColumnas) + columnas1] = colorAux2;
		dev_tablero[(fila2*tamColumnas) + columna2] = colorAux1;

		int sameVertical1 = comprobarIgualesPos(dev_tablero, filas1, columnas1, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, filas1, columnas1, arriba, tamFilas, tamColumnas);
		int sameHorizon1 = comprobarIgualesPos(dev_tablero, filas1, columnas1, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, filas1, columnas1, izquierda, tamFilas, tamColumnas);
		int sameVertical2 = comprobarIgualesPos(dev_tablero, fila2, columna2, abajo, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, arriba, tamFilas, tamColumnas);
		int sameHorizon2 = comprobarIgualesPos(dev_tablero, fila2, columna2, derecha, tamFilas, tamColumnas) + comprobarIgualesPos(dev_tablero, fila2, columna2, izquierda, tamFilas, tamColumnas);

		if (sameVertical1 >= 2 || sameHorizon1 >= 2) { //Comprobamos que en cualquiera de las posiciones haya bombas que puedan explotar
			explotan = true;
		}
		else if (sameVertical2 >= 2 || sameHorizon2 >= 2) {
			explotan = true;
		}

		// deshacemos los cambios en la matriz
		dev_tablero[(filas1*tamColumnas) + columnas1] = colorAux1;
		dev_tablero[(fila2*tamColumnas) + columna2] = colorAux2;
	
	return explotan;
	}
}


__global__ void probeMovPosi(int* dev_tablero, int filas1, int columnas1, int fila2, int columna2,int tamFilas,int tamColumnas,char modoJuego,bool* dev_mov, int* dev_contadorEliminados) {
	int fil = blockIdx.y * blockDim.y + threadIdx.y;		//Indice de la x
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	bool TrueMov;
	bool explot;
		if ((col == columnas1 && fil == filas1)) {//comprobamos que el hilo que accede a la funcion sea el que queremos cambiar(se comprueban en las funciones los dos numeros)
			TrueMov = adyacentes(filas1, columnas1, fila2, columna2);
			explot = explotChange(dev_tablero, filas1, columnas1, fila2, columna2, tamFilas, tamColumnas);
		if (TrueMov && explot) {
				//Si ambos son true realizamos el cmbio
				int colorAux1 = dev_tablero[(filas1*tamColumnas) + columnas1];
				int colorAux2 = dev_tablero[(fila2*tamColumnas) + columna2];
				dev_tablero[(filas1*tamColumnas) + columnas1] = colorAux2;
				dev_tablero[(fila2*tamColumnas) + columna2] = colorAux1;
				*dev_mov = true;
			}
			else { printf("MOVIMIENTO ERRONEO, Las posiciones no son adyacentes o no explotan\n"); }
		}

}

hipError_t jugar(int* tablero, int tamFilas, int tamColumnas, int* contadorEliminados, char m, int nColores) {

	hipError_t cudaStatus;
	int fila1 = 0, columna1 = 0, fila2 = 0, columna2 = 0;
	bool* mov = false, *dev_mov;
	int* dev_tablero;
	int *dev_contadorEliminados;
	int bomba = 0;
	int explota = 0;
	bool hayBomba = false;
	char guardar=NULL;

	dim3 blocks(1);
	dim3 threads(tamFilas, tamColumnas);
	//Asignamos objeto a memoria global con cudamalloc
	cudaStatus = hipMalloc((void**)&dev_tablero, tamFilas*tamColumnas * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_contadorEliminados, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_mov, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Pasamos parametros a la parte del device
	cudaStatus = hipMemcpy(dev_tablero, tablero, tamFilas*tamColumnas * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_tablero failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_contadorEliminados, contadorEliminados, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_contadorEliminados failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mov, &mov, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_mov failed!");
		goto Error;
	}
	bool activada = false;
	while (!hasMoreMovements(tablero, tamFilas, tamColumnas) && !activada) {
		
		printf("NO HAY MOVIMIENTO\n\nIntroduce el numero de la bomba: ");
		scanf("%d", &bomba);
			if (bomba == 91) {
				printf("Introduce el numero de la fila que deseas explotar:");
				scanf("%d", &explota);
				activada = true;
			}
			if (bomba == 92) {
				printf("Introduce el numero de la columna que deseas explotar:");
				scanf("%d", &explota);
				activada = true;
			}
			if (bomba == 93) {
				explota = 0;
				activada = true;
			}
			menuBombas << <blocks, threads >> > (dev_tablero, tamFilas, tamColumnas, explota, bomba);
			hayBomba = true;

	}
	//Modo manual
	if (m == 'm' && hasMoreMovements(tablero, tamFilas, tamColumnas)) {
		
		printf("-Para usar una bomba introduce 90 ");
		printf("\n-Para terminar el juego 99 ");
		printf("\nIntroduzca la fila del primer diamante: ");
		scanf_s("%d", &fila1);
		if (fila1 == 99) {
			printf("\nQuieres guardar la partida? (s/n): ");
			getchar();
			
				guardarPartida(tablero, tamFilas, tamColumnas, nColores);
				printf("\nPartida guardada correctamente");
		
			
			printf("\n\n-JUEGO TERMINADO-");
			Sleep(1000);
			exit(0);
		}
			if (fila1 == 90) {
				printf("\nIntroduce el numero de la bomba:");
				scanf("%d", &bomba);
				if (bomba == 91) {
					printf("Introduce el numero de la fila que deseas explotar:");
					scanf("%d", &explota);
				}
				if (bomba == 92) {
					printf("Introduce el numero de la columna que deseas explotar:");
					scanf("%d", &explota);
				}
				if (bomba == 93) explota = 0;
				menuBombas << <blocks, threads >> > (dev_tablero, tamFilas, tamColumnas, explota, bomba);
				hayBomba = true;
			}else {
				printf("Introduzca la columna del primer diamante: ");
				scanf_s("%d", &columna1);
				printf("Introduzca la fila del segundo diamante: ");
				scanf_s("%d", &fila2);
				printf("Introduzca la columna del segundo diamante: ");
				scanf_s("%d", &columna2);

				//bool canMove = hasMoreMovements(tablero);
			}
			
	}
	else if(hasMoreMovements(tablero, tamFilas, tamColumnas)){
		
		int posX = 0;
		int movOptimoFila1, movOptimoColumna1, movOptimoFila2, movOptimoColumna2, contMovOptimo = 0;
		int contDiamantesExplot = 0;
		while (posX < tamFilas) {
			for (int posY = 0; posY < tamColumnas; posY++) {
				if (posX + 1 < tamFilas && explotan(tablero, posX, posY, posX + 1, posY, tamFilas, tamColumnas)) { // Abajo
					contDiamantesExplot = autoContMov(tablero, posX, posY, posX + 1, posY, tamFilas, tamColumnas);
					if (contDiamantesExplot >= contMovOptimo) {
						contMovOptimo = contDiamantesExplot;
						movOptimoFila1 = posX;
						movOptimoColumna1 = posY;
						movOptimoFila2 = posX + 1;
						movOptimoColumna2 = posY;
					}
				}
				else if (posY + 1 < tamColumnas && explotan(tablero, posX, posY, posX, posY + 1, tamFilas, tamColumnas)) { //Derecha
					contDiamantesExplot = autoContMov(tablero, posX, posY, posX, posY + 1, tamFilas, tamColumnas);
					if (contDiamantesExplot >= contMovOptimo) {
						contMovOptimo = contDiamantesExplot;
						movOptimoFila1 = posX;
						movOptimoColumna1 = posY;
						movOptimoFila2 = posX;
						movOptimoColumna2 = posY + 1;
					}
				}
				else if (posY - 1 >= 0 && explotan(tablero, posX, posY, posX, posY - 1, tamFilas, tamColumnas)) { //Izquierda
					contDiamantesExplot = autoContMov(tablero, posX, posY, posX, posY - 1, tamFilas, tamColumnas);
					if (contDiamantesExplot > contMovOptimo) {
						contMovOptimo = contDiamantesExplot;
						movOptimoFila1 = posX;
						movOptimoColumna1 = posY;
						movOptimoFila2 = posX;
						movOptimoColumna2 = posY - 1;
					}
				}
				else if (posX - 1 >= 0 && explotan(tablero, posX, posY, posX - 1, posY, tamFilas, tamColumnas)) {//Arriba
					contDiamantesExplot = autoContMov(tablero, posX, posY, posX - 1, posY, tamFilas, tamColumnas);
					if (contDiamantesExplot >= contMovOptimo) {
						contMovOptimo = contDiamantesExplot;
						movOptimoFila1 = posX;
						movOptimoColumna1 = posY;
						movOptimoFila2 = posX - 1;
						movOptimoColumna2 = posY;
					}
				}
			}
			posX++;
		}
			//Definimos el movimiento mas optimo para explotar
			fila1 = movOptimoFila1; fila2 = movOptimoFila2;
			columna1 = movOptimoColumna1; columna2 = movOptimoColumna2;
			printf("Movimiento mas optimo: \n Fila: %d Columna: %d \n Fila: %d Columna: %d ", fila1, columna1, fila2, columna2);
		}


	if (!hayBomba && hasMoreMovements(tablero, tamFilas, tamColumnas))
	{
		probeMovPosi << <blocks, threads >> > (dev_tablero, fila1, columna1, fila2, columna2, tamFilas, tamColumnas, m, dev_mov,dev_contadorEliminados);
		cudaStatus = hipMemcpy(&mov, dev_mov, sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy device to host dev_mov failed!");
			goto Error;
		}

	}
	
	if (mov && !hayBomba && hasMoreMovements(tablero, tamFilas, tamColumnas)) {
		jugarKernel << <blocks, threads >> >(dev_tablero, fila1, columna1, fila2, columna2, tamFilas, tamColumnas, dev_contadorEliminados);
	}
	

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "jugarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(tablero, dev_tablero, tamFilas*tamColumnas * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy tablero failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(contadorEliminados, dev_contadorEliminados, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy contadorEliminados failed!");
		goto Error;
	}

Error:
	hipFree(dev_tablero);
	hipFree(dev_contadorEliminados);
	hipFree(dev_mov);

	return cudaStatus;

}

//Metodo para guardar la partida en txt
void guardarPartida(int* tablero, int filas, int columnas, int dificultad) {

	doc = fopen("guardado.txt", "w");
	
		fprintf(doc, "%i \n", filas);
		fprintf(doc, "%i \n", columnas);
		fprintf(doc, "%i \n", dificultad);
		for (int i = 0; i < (filas*columnas); i++) {
			fprintf(doc, "%i ", tablero[i]);
		}
		fclose(doc);
}

//Funcion para cargar una partida guardada en el txt guardado
void cargarPartida() {
	
	hipError_t cudaStatus;
	leer = fopen("guardado.txt", "r");
	int filas=0;
	int columnas=0;
	int dificultad=0;
	int contadorEliminados = 0;

	// leer el variables del txt
	fscanf(leer, "%d", &filas);
	printf("FILAS: %d",filas);
	
	fscanf(leer, "%d", &columnas);
	printf("\nCOLUMNAS: %d", columnas);

	fscanf(leer, "%d", &dificultad);
	printf("\nCOLORES: %d", dificultad);
	
	int* tablero = (int*)malloc(filas*columnas * sizeof(int));
	
	for (int i = 0; i < filas*columnas; i++) {
		fscanf(leer, "%d", &tablero[i]);
	}

	do {

		imprimeTablero(tablero, filas, columnas);
		cudaStatus = jugar(tablero, filas, columnas, &contadorEliminados, 'm', dificultad);

		imprimeTablero(tablero, filas, dificultad);
		tablero = rellenarTablero(tablero, filas, columnas, dificultad);
		printf("Contador  = %d\n ", contadorEliminados);

	} while ((cudaStatus == 0) && (contadorEliminados < 100));
}
	
int main() {
	//Declaracion de variables para la ejecucion del programa

	hipError_t cudaStatus;
	int tamFilas; //Filas que tendra el tablero del programa
	int tamColumnas; //Columnas que tendra el tablero del programa
	char modo; //Modo de ejecucion del programa
	int contadorEliminados = 0;
	char dificultad;
	int* tablero;
	int nColores;
	char cargar;

	printf("\nQuieres cargar una partida? (s/n): ");
	scanf("%c",&cargar);
	getchar();
	if(cargar=='s') cargarPartida();
	else {
		modo = pedirModoEjecucion();
		dificultad = pedirDificultad();
		tamFilas = pedirFilasTablero();
		tamColumnas = pedirColumnasTablero();

		if (dificultad == 'F') {
			nColores = 4;
		}
		else if (dificultad == 'M') {
			nColores = 6;
		}
		else {
			nColores = 8;
		}

		printf("\nLos datos introducidos por el usuario son: -%c %c %d %d\n", modo, dificultad, tamFilas, tamColumnas);
		cudaStatus = comprobarPropiedades(tamFilas, tamColumnas);

		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		tablero = generaTablero(tamFilas, tamColumnas, nColores);

	}
	do {
		
		imprimeTablero(tablero, tamFilas, tamColumnas);

		if (modo == 'm') {
			cudaStatus = jugar(tablero, tamFilas, tamColumnas, &contadorEliminados, 'm',nColores);
			
		}
		else
		{

			cudaStatus = jugar(tablero, tamFilas, tamColumnas, &contadorEliminados, 'a',nColores);
			getchar();
		}
	

		imprimeTablero(tablero, tamFilas, tamColumnas);
		tablero = rellenarTablero(tablero, tamFilas, tamColumnas, nColores);
		printf("Contador  = %d\n ", contadorEliminados);

	} while ((cudaStatus == 0) && (contadorEliminados < 100));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Algo ha fallado!");
		goto Error;
	}

	printf(" - - - - - - JUEGO TERMINADO - - - - - - - ");
	
Error:

	getchar();
	getchar();


	return cudaStatus;
}

int* rellenarTablero(int* tablero, int tamFilas, int tamColumnas, int nColores) {
	for (int i = 0; i < tamFilas*tamColumnas; i++)
	{
		if(tablero[i] == 0) tablero[i] = 1 + (rand() % nColores);
	}
	return tablero;
}
//Procedimiento que imprime una matriz de filas * columnas de enteros indicando fila y columna 
void imprimeTablero(int* tablero, int filas, int columnas) {

	printf("\n     ------TABLERO------\n\n      ");

	//Imprime el numero de columna
	for (int i = 0; i < columnas; i++) {

		if (i > 99)printf("%d ", i);
		else if (i > 9) printf("%d  ", i);
		else printf("%d   ", i);

	}

	printf("\n");
	for (int i = 0; i < filas*columnas; i++) {

		//Imprime el numero de fila
		if (i % columnas == 0) {

			printf("\n");

			if ((i / columnas) > 99)printf("%d | ", i / columnas);
			else if ((i / columnas) > 9) printf("%d  | ", i / columnas);
			else printf("%d   | ", i / columnas);


		}
		//Según el valor en la posición i del tablero se imprime de un color u otro
		switch (tablero[i]) {
		case 0: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 0);
			break;
		case 1: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1);
			break;
		case 2: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 2);
			break;
		case 3: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 12);
			break;
		case 4: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13);
			break;
		case 5: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14);
			break;
		case 6: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
			break;
		case 7: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 3);
			break;
		case 8: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 10);
			break;
		case 9: SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 11);
			break;
		default:;
		}
		printf("%d", tablero[i]);

		//Se imprime de nuevo en blanco
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 7);
		printf(" | ");

	}


	printf("\n\n");

}


/*
Procedimiento que genera un tablero de size x size relleno con numeros del 1 al 2
y con bombas representadas con el número 3
*/
int* generaTablero(int filas, int columnas, int nColores) {

	//Reserva de memoria para el numero de columnas
	int* tablero = (int*)malloc(filas*columnas * sizeof(int));

	for (int i = 0; i < filas*columnas; i++)
	{

			tablero[i] = 1 + (rand() % nColores);

	}
	return tablero;

}

char pedirDificultad() {
	char dificultad=' ';
	getchar();
	while (dificultad != 'F' && dificultad != 'M' && dificultad != 'D') {
		printf("Que dificultad desea para el juego? Facil(F), Medio(M), Dificil(D)\n");
		fflush(stdin);
		scanf("%c", &dificultad);
		if (dificultad != 'F' && dificultad != 'M' && dificultad != 'D' ) {
			printf("Usted ha introducido una dificutad no existente: -%c.\n", dificultad);
			printf("Por favor, introduzca uno de las dificultades que se le presentan a continuacion.\n\n");
		}

	};
	
	return dificultad;

}

//Metodo que solicita al usuario el modo de ejecucion del programa
char pedirModoEjecucion() {
	char modo;
	do {
		printf("Existen 2 modos de ejecucion para Jewels Legend:\n\n");
		printf("- Automatica(a): el programa pulsara aleatoriamente las teclas del tablero\n");
		printf("- Manual(m): el programa esperara a que el usuario pulse las teclas del tablero\n");
		printf("Introduce el modo de ejecucion del programa: "); 
		fflush(stdin);
		scanf("%c", &modo);
		if (modo != 'a' && modo != 'm') {
			printf("Usted ha introducido un modo de ejecucion no existente: %c.\n", modo);
			printf("Por favor, introduzca uno de los modos que se le presentan a continuacion.\n\n");
		}
	} while (modo != 'a' && modo != 'm');
	return modo;
}

//Metodo que solicita al usuario el numero de filas que tendra el tablero
int pedirFilasTablero() {
	int filas;

	do {
		printf("\nIntroduzca las filas que tendra el tablero: ");
		fflush(stdin);
		scanf_s("%d", &filas);
		if (filas < 1 || filas > 2147483647) {
			printf("Introduzca un numero de filas correcto\n");
		}
	} while (filas < 1 || filas > 2147483647); //El numero de filas tiene que ser un numero entero positivo
	return filas;
}

//Metodo que solicita al usuario el numero de columnas que tendra el tablero
int pedirColumnasTablero() {
	int columnas;
	do {
		printf("\nIntroduzca las columnas que tendra el tablero: ");
		fflush(stdin);
		scanf_s("%d", &columnas);
		if (columnas < 1 || columnas > 2147483647) {
			printf("Introduzca un numero de columnas correcto\n");
		}
	} while (columnas < 1 || columnas > 2147483647); //El numero de filas tiene que ser un numero entero positivo
	return columnas;
}

void prop() {

	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execition timeout : ");



		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %lu\n", prop.totalGlobalMem);
		printf("Total constant Mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n",
			prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n",
			prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);

		printf("\n");
	}

}